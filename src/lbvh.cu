#include "hip/hip_runtime.h"
#include "lbvh.h"

// This optimized LBVH is based on the paper "Maximizing Parallelism in the Construction of BVHs,
// Octrees, and k-d Trees" by Tero Karras of NVIDIA Research

bool morton_sort(const MortonCode& a, const MortonCode& b) {
    return a.code < b.code;
}

bool isLeaf(const LBVHNode* node) {
    return node->left == 0xFFFFFFFF && node->right == 0xFFFFFFFF;
}

AABB Union(AABB left, AABB right) {
    glm::vec3 umin = glm::min(left.min, right.min);
    glm::vec3 umax = glm::max(left.max, right.max);
    return AABB{ umin, umax }; 
}

// Expand 10-bit integer into 30-bit integer
unsigned int expand(unsigned int n)
{
    n = (n | (n << 16)) & 0b00000011000000000000000011111111;
    n = (n | (n << 8)) & 0b00000011000000001111000000001111;
    n = (n | (n << 4)) & 0b00000011000011000011000011000011;
    n = (n | (n << 2)) & 0b00001001001001001001001001001001;
    return n;
}

// Based on PBRT 4.3.3. and Tero Karras version at https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
unsigned int mortonCode3D(const glm::vec3& centroid) {
    // Convert centroid coordinates to value between 0 and 1024
    float x = min(max(centroid.x * 1024.0f, 0.0f), 1023.0f);
    float y = min(max(centroid.y * 1024.0f, 0.0f), 1023.0f);
    float z = min(max(centroid.z * 1024.0f, 0.0f), 1023.0f);

    // Expand each 10 bit value so that ith value is at 3 * ith position
    unsigned int xx = expand((unsigned int)x);
    unsigned int yy = expand((unsigned int)y);
    unsigned int zz = expand((unsigned int)z);

    // Interleave the bits
    return (xx << 2) | (yy << 1) | zz;
}

void computeMortonCodes(Scene* scene, const AABB& sceneAABB) {
    for (int i = 0; i < scene->triangles.size(); i++) {
        // Find centroid of triangle's bounding box
        glm::vec3 centroid = 0.5f * scene->triangles[i].aabb.min + 0.5f * scene->triangles[i].aabb.max;

        // Normalize centroid w.r.t. scene bounding box
        glm::vec3 norm_centroid = (centroid - sceneAABB.min) / (sceneAABB.max - sceneAABB.min);

        // Calculate Morton code and add to list
        MortonCode mcode;
        mcode.objectId = i;
        mcode.code = mortonCode3D(norm_centroid);
        scene->mcodes.push_back(mcode);
    }
}

void sortMortonCodes(Scene* scene) {
    std::vector<MortonCode> mcodes_copy = scene->mcodes;
    std::sort(mcodes_copy.begin(), mcodes_copy.end(), morton_sort);
    scene->mcodes = mcodes_copy;
}

// Determines the number of common bits between two numbers 
int delta(MortonCode* sortedMCodes, int N, int i, int j) {
    // Range check
    if (j < 0 || j >= N) {
        return -1;
    }

    if (sortedMCodes[i].code == sortedMCodes[j].code)
    {
        return 32 + __lzcnt(i ^ j);
    }
    
    return __lzcnt(sortedMCodes[i].code ^ sortedMCodes[j].code);
}

// Determines in which direction the node's range will grow
int sign(MortonCode* sortedMCodes, int N, int i) {
    int diff = delta(sortedMCodes, N, i, i + 1) - delta(sortedMCodes, N, i, i - 1);
    return (diff >= 0) ? 1 : -1;
}

NodeRange determineRange(MortonCode* sortedMCodes, int triangleCount, int i) {
    // Determine direction of range (+1 or -1)
    int d = sign(sortedMCodes, triangleCount, i);

    // Compute upper bound of range
    int deltaMin = delta(sortedMCodes, triangleCount, i, i - d);
    int lMax = 2;
    while (delta(sortedMCodes, triangleCount, i, i + lMax * d) > deltaMin) {
        lMax = lMax * 2;
    }

    // Find the other end with binary search
    int l = 0;
    for (int t = lMax / 2; t >= 1; t /= 2) {
        if (delta(sortedMCodes, triangleCount, i, i + (l + t) * d) > deltaMin) {
            l = l + t;
        }
    }
    int j = i + l * d;

    return NodeRange{ i, j, l, d };
}

int findSplit(MortonCode* sortedMCodes, int triangleCount, NodeRange range) {
    int i = range.i;
    int j = range.j;
    int l = range.l;
    int d = range.d;    
    
    // Find split position with binary search
    int deltaNode = delta(sortedMCodes, triangleCount, range.i, range.j);
    int s = 0;
    int t = l;
    do {
        t = ceil(t / 2.f);
        if (delta(sortedMCodes, triangleCount, i, i + (s + t) * d) > deltaNode) {
            s = s + t;
        }
    } while (t > 1);

    int gamma = i + s * d + min(d, 0);
    
    return gamma;
}

// Recursively assigns bounding boxes to each node, start from the leaf nodes and recursing upwards
AABB assignBoundingBoxes(Scene* scene, LBVHNode* node) {

    if (!isLeaf(node)) {
        AABB leftAABB = assignBoundingBoxes(scene, &scene->lbvh[node->left]);
        AABB rightAABB = assignBoundingBoxes(scene, &scene->lbvh[node->right]);
        node->aabb = Union(leftAABB, rightAABB);
    }

    return node->aabb;
}

// Tree-building functions
void buildLBVH(Scene* scene, int leafStart, int triangleCount, int meshNum) {
    // Resize LBVH
    int numLeaf = triangleCount;
    int numInternal = triangleCount - 1;
    int internalStart = leafStart + numLeaf;
    scene->lbvh.resize(numLeaf + numInternal);
    scene->sorted_triangles.resize(numLeaf);

    // Initialize leaf nodes
    for (int i = leafStart; i < numLeaf; ++i) {
        LBVHNode leafNode;
        leafNode.objectId = scene->mcodes[i - leafStart].objectId; 
        leafNode.aabb = scene->triangles[leafNode.objectId].aabb;
        leafNode.left = 0xFFFFFFFF;
        leafNode.right = 0xFFFFFFFF;
        scene->lbvh[i] = leafNode;

        scene->sorted_triangles[i] = scene->triangles[leafNode.objectId];
    }
    scene->triangles = scene->sorted_triangles;

    // Initialize internal nodes
    for (int j = internalStart; j < internalStart + numInternal; ++j) {
        LBVHNode internalNode;

        // Determine range
        NodeRange range = determineRange(scene->mcodes.data(), triangleCount, j - triangleCount);

        // Find split position
        int split = findSplit(scene->mcodes.data(), triangleCount, range);
    
        int leftChild = -1;
        int rightChild = -1;
        if (min(range.i, range.j) == split) {
            leftChild = split;
        }
        else {
            leftChild = triangleCount + split;
        }

        if (max(range.i, range.j) == split + 1) {
            rightChild = split + 1;
        }
        else {
            rightChild = triangleCount + split + 1;
        }

        internalNode.objectId = -1;
        internalNode.left = leftChild;
        internalNode.right = rightChild;
        scene->lbvh[j] = internalNode;
    }
    // Assign bounding boxes here
    assignBoundingBoxes(scene, &scene->lbvh[triangleCount]);
}

void generateLBVH(Scene* scene)
{
    for (int i = 0; i < scene->meshCount; i++) {
        // Morton code computation
        computeMortonCodes(scene, scene->mesh_aabbs[i]);

        // Sort Morton codes
        sortMortonCodes(scene);

        // Build tree from sorted Morton codes
        buildLBVH(scene, scene->lbvh.size(), scene->mcodes.size(), i);

        scene->mcodes.clear();
    }
}