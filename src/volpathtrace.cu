#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "volPathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define BLOCK_SIZE_1D 128
#define BLOCK_SIZE_2D 16

#define MIN_INTERSECT_DIST 0.0001f
#define MAX_INTERSECT_DIST 10000.0f

#define ENABLE_RECTS
#define ENABLE_SPHERES
#define ENABLE_TRIS
#define ENABLE_SQUAREPLANES



#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn_Vol(msg, FILENAME, __LINE__)
void checkCUDAErrorFn_Vol(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine_Vol(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Tri* dev_tris = NULL;
static Light* dev_lights = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static BVHNode_GPU* dev_bvh_nodes = NULL;
static Medium* dev_media = NULL;
static nanovdb::NanoGrid<float>* dev_media_density = NULL;
//hipStream_t media_stream;

static MISLightRay* dev_direct_light_rays = NULL;
static MISLightIntersection* dev_direct_light_isects = NULL;

static MISLightRay* dev_bsdf_light_rays = NULL;
static MISLightIntersection* dev_bsdf_light_isects = NULL;

static glm::vec3* dev_sample_colors = NULL;

int pixelcount_vol;

void InitDataContainer_Vol(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

// TODO: remove these when done testing
__global__ void grid_test_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
	if (threadIdx.x > 6)
		return;
	int i = 97 + threadIdx.x;
	auto gpuAcc = deviceGrid->getAccessor();
	printf("(%3i,0,0) NanoVDB gpu: % -4.2f\n", i, gpuAcc.getValue(nanovdb::Coord(i, i, i)));
}

void volPathtraceInit(Scene* scene) {

	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	pixelcount_vol = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount_vol * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount_vol * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount_vol * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_tris, scene->num_tris * sizeof(Tri));
	hipMemcpy(dev_tris, scene->mesh_tris_sorted.data(), scene->num_tris * sizeof(Tri), hipMemcpyHostToDevice);

	hipMalloc(&dev_bvh_nodes, scene->bvh_nodes_gpu.size() * sizeof(BVHNode_GPU));
	hipMemcpy(dev_bvh_nodes, scene->bvh_nodes_gpu.data(), scene->bvh_nodes_gpu.size() * sizeof(BVHNode_GPU), hipMemcpyHostToDevice);

	hipMalloc(&dev_lights, scene->lights.size() * sizeof(Light));
	hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Light), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount_vol * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount_vol * sizeof(ShadeableIntersection));

	hipMalloc(&dev_media, scene->media.size() * sizeof(Medium));
	hipMemcpy(dev_media, scene->media.data(), scene->media.size() * sizeof(Medium), hipMemcpyHostToDevice);

	// Copy NanoVDB grid to the GPU
	scene->gridHandle.deviceUpload();
	dev_media_density = scene->gridHandle.deviceGrid<float>();
	//grid_test_kernel <<< 1, 64 >>> (dev_media_density);
	
	// Copy NanoVDB grid to the GPU asynchronously (for later)
	//hipStreamCreate(&media_stream);
	//scene->gridHandle.deviceUpload(media_stream, false);
	
	// FOR LIGHT SAMPLED MIS RAY
	hipMalloc(&dev_direct_light_rays, pixelcount_vol * sizeof(MISLightRay));

	hipMalloc(&dev_direct_light_isects, pixelcount_vol * sizeof(MISLightIntersection));
	hipMemset(dev_direct_light_isects, 0, pixelcount_vol * sizeof(MISLightIntersection));

	// FOR BSDF SAMPLED MIS RAY
	hipMalloc(&dev_bsdf_light_rays, pixelcount_vol * sizeof(MISLightRay));

	hipMalloc(&dev_bsdf_light_isects, pixelcount_vol * sizeof(MISLightIntersection));
	hipMemset(dev_bsdf_light_isects, 0, pixelcount_vol * sizeof(MISLightIntersection));

	// TODO: initialize any extra device memeory you need

}

void volPathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_tris);
	hipFree(dev_bvh_nodes);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_lights);
	hipFree(dev_direct_light_rays);
	hipFree(dev_direct_light_isects);
	hipFree(dev_bsdf_light_rays);
	hipFree(dev_bsdf_light_isects);
}

__global__ void generateRayFromThinLensCamera_Vol(Camera cam, int iter, int traceDepth, float jitterX, float jitterY, glm::vec3 thinLensCamOrigin, glm::vec3 newRef,
	PathSegment* pathSegments)
{
	__shared__ PathSegment mat[BLOCK_SIZE_2D][BLOCK_SIZE_2D];

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);

	if (x < cam.resolution.x && y < cam.resolution.y) {
		mat[threadIdx.x][threadIdx.y] = pathSegments[index];
		PathSegment& segment = mat[threadIdx.x][threadIdx.y];

		segment.ray.origin = thinLensCamOrigin;
		segment.rng_engine = makeSeededRandomEngine_Vol(iter, index, traceDepth);
		segment.rayThroughput = glm::vec3(1.0f, 1.0f, 1.0f);
		segment.accumulatedIrradiance = glm::vec3(0.0f, 0.0f, 0.0f);
		segment.prev_hit_was_specular = false;
		segment.prev_hit_null_material = false;

		float jittered_x = ((float)x) + jitterX;
		float jittered_y = ((float)y) + jitterY;

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(
			glm::normalize(newRef - thinLensCamOrigin) - cam.right * cam.pixelLength.x * (jittered_x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * (jittered_y - (float)cam.resolution.y * 0.5f)
		);

		segment.ray.direction_inv = 1.0f / segment.ray.direction;

		segment.remainingBounces = traceDepth;

		pathSegments[index] = mat[threadIdx.x][threadIdx.y];
	}
}

__global__ void generateRayFromCamera_Vol(Camera cam, int iter, int traceDepth, float jitterX, float jitterY,
	PathSegment* pathSegments)
{
	__shared__ PathSegment mat[BLOCK_SIZE_2D][BLOCK_SIZE_2D];

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);

	if (x < cam.resolution.x && y < cam.resolution.y) {
		mat[threadIdx.x][threadIdx.y] = pathSegments[index];
		PathSegment& segment = mat[threadIdx.x][threadIdx.y];

		segment.ray.origin = cam.position;
		segment.rng_engine = makeSeededRandomEngine_Vol(iter, index, traceDepth);
		segment.rayThroughput = glm::vec3(1.0f, 1.0f, 1.0f);
		segment.accumulatedIrradiance = glm::vec3(0.0f, 0.0f, 0.0f);
		segment.prev_hit_was_specular = false;
		segment.prev_hit_null_material = false;
		segment.medium = cam.medium;

		float jittered_x = ((float)x) + jitterX;
		float jittered_y = ((float)y) + jitterY;

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(
			cam.view - cam.right * cam.pixelLength.x * (jittered_x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * (jittered_y - (float)cam.resolution.y * 0.5f)
		);

		segment.ray.direction_inv = 1.0f / segment.ray.direction;

		segment.remainingBounces = traceDepth;

		pathSegments[index] = mat[threadIdx.x][threadIdx.y];
	}
}

__global__ void computeIntersections_Vol(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, Tri* tris
	, int tris_size
	, ShadeableIntersection* intersections
	, BVHNode_GPU* bvh_nodes
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		if (pathSegments[path_index].remainingBounces == 0) {
			return;
		}
		Ray r = pathSegments[path_index].ray;

		ShadeableIntersection isect;
		isect.t = MAX_INTERSECT_DIST;

		float t;


		glm::vec3 tmp_normal;
		int obj_ID = -1;

#ifdef ENABLE_TRIS
		if (tris_size != 0) {
			int stack_pointer = 0;
			int cur_node_index = 0;
			int node_stack[32];
			BVHNode_GPU cur_node;
			glm::vec3 P;
			glm::vec3 s;
			float t1;
			float t2;
			float tmin;
			float tmax;
			while (true) {
				cur_node = bvh_nodes[cur_node_index];

				// (ray-aabb test node)
				t1 = (cur_node.AABB_min.x - r.origin.x) * r.direction_inv.x;
				t2 = (cur_node.AABB_max.x - r.origin.x) * r.direction_inv.x;

				tmin = glm::min(t1, t2);
				tmax = glm::max(t1, t2);

				t1 = (cur_node.AABB_min.y - r.origin.y) * r.direction_inv.y;
				t2 = (cur_node.AABB_max.y - r.origin.y) * r.direction_inv.y;

				tmin = glm::max(tmin, glm::min(t1, t2));
				tmax = glm::min(tmax, glm::max(t1, t2));

				t1 = (cur_node.AABB_min.z - r.origin.z) * r.direction_inv.z;
				t2 = (cur_node.AABB_max.z - r.origin.z) * r.direction_inv.z;

				tmin = glm::max(tmin, glm::min(t1, t2));
				tmax = glm::min(tmax, glm::max(t1, t2));

				if (tmax >= tmin) {
					// we intersected AABB
					if (cur_node.tri_index != -1) {
						// this is leaf node
						// triangle intersection test
						Tri tri = tris[cur_node.tri_index];

						t = glm::dot(tri.plane_normal, (tri.p0 - r.origin)) / glm::dot(tri.plane_normal, r.direction);
						if (t >= -0.0001f) {
							P = r.origin + t * r.direction;

							// barycentric coords
							s = glm::vec3(glm::length(glm::cross(P - tri.p1, P - tri.p2)),
								glm::length(glm::cross(P - tri.p2, P - tri.p0)),
								glm::length(glm::cross(P - tri.p0, P - tri.p1))) / tri.S;

							if (s.x >= -0.0001f && s.x <= 1.0001f && s.y >= -0.0001f && s.y <= 1.0001f &&
								s.z >= -0.0001f && s.z <= 1.0001f && (s.x + s.y + s.z <= 1.0001f) && (s.x + s.y + s.z >= -0.0001f) && isect.t > t) {
								isect.t = t;
								isect.materialId = tri.mat_ID;
								isect.surfaceNormal = glm::normalize(s.x * tri.n0 + s.y * tri.n1 + s.z * tri.n2);

								// Check if surface is medium transition
								if (IsMediumTransition(tri.mediumInterface)) {
									isect.mediumInterface = tri.mediumInterface;
								}
								else {
									MediumInterface mediumInterface;
									mediumInterface.inside = pathSegments[path_index].medium;
									mediumInterface.outside = pathSegments[path_index].medium;
									isect.mediumInterface = mediumInterface;
								}
							}
						}
						// if last node in tree, we are done
						if (stack_pointer == 0) {
							break;
						}
						// otherwise need to check rest of the things in the stack
						stack_pointer--;
						cur_node_index = node_stack[stack_pointer];
					}
					else {	
						node_stack[stack_pointer] = cur_node.offset_to_second_child;
						stack_pointer++;
						cur_node_index++;
					}
				}
				else {
					// didn't intersect AABB, remove from stack
					if (stack_pointer == 0) {
						break;
					}
					stack_pointer--;
					cur_node_index = node_stack[stack_pointer];
				}
			}
	}
#endif


		for (int i = 0; i < geoms_size; ++i)
		{
			Geom& geom = geoms[i];



			if (geom.type == SPHERE) {
#ifdef ENABLE_SPHERES
				t = sphereIntersectionTest(geom, r, tmp_normal);
#endif                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                 
			}
			else if (geom.type == SQUAREPLANE) {
#ifdef ENABLE_SQUAREPLANES
				t = squareplaneIntersectionTest(geom, r, tmp_normal);
#endif	
			}
			else {
#ifdef ENABLE_RECTS
			t = boxIntersectionTest(geom, r, tmp_normal);
#endif
			}

			if (depth == 0 && glm::dot(tmp_normal, r.direction) > 0.0) { 
				continue; 
			}
			else if (isect.t > t) {
				isect.t = t;
				isect.materialId = geom.materialid;
				isect.surfaceNormal = tmp_normal;

				// Check if surface is medium transition
				if (IsMediumTransition(geom.mediumInterface)) {
					isect.mediumInterface = geom.mediumInterface;
				}
				else {
					isect.mediumInterface.inside = pathSegments[path_index].medium;
					isect.mediumInterface.outside = pathSegments[path_index].medium;
				}
			}
			
		}

		if (isect.t >= MAX_INTERSECT_DIST) {
			// hits nothing
			pathSegments[path_index].remainingBounces = 0;
		}
		else {
			intersections[path_index] = isect;
		}
	}
}

__global__ void sampleParticipatingMedium(
	int num_paths,
	PathSegment* pathSegments,
	ShadeableIntersection* intersections,
	Medium* media,
	const nanovdb::NanoGrid<float>* media_density
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}

		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

		int rayMediumIndex = pathSegments[idx].medium;
		MediumInteraction mi;
		mi.medium = -1;
		if (rayMediumIndex >= 0) {
			if (media[rayMediumIndex].type == HOMOGENEOUS) {
				pathSegments[idx].rayThroughput *= Sample_homogeneous(media[rayMediumIndex], pathSegments[idx], intersections[idx], &mi, rayMediumIndex, u01(rng));
			}
			else {
				pathSegments[idx].rayThroughput *= Sample_heterogeneous(media[rayMediumIndex], pathSegments[idx], intersections[idx], &mi, media_density, rayMediumIndex, rng, u01);
			}
		}
		if (glm::length(pathSegments[idx].rayThroughput) == 0.0f) {
			pathSegments[idx].remainingBounces = 0;
		}
		intersections[idx].mi = mi;
	}
}

// kernel to handle interactions within a medium (instead of surface)
__global__ void generateMediumDirectLightSample(
	int num_paths,
	int max_depth,
	PathSegment* pathSegments,
	Material* materials,
	ShadeableIntersection* intersections,
	Medium* media,
	MISLightRay* direct_light_rays,
	MISLightIntersection* direct_light_isects,
	Light* lights,
	int num_lights,
	Geom* geoms
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}
		if (intersections[idx].mi.medium == -1) {
			return;
		}

		ShadeableIntersection intersection = intersections[idx];
		Material material = materials[intersection.materialId];

		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

		// TODO: Check surface intersection bsdf doesn't exist
		computeDirectLightSamplePreVis(
			idx,
			pathSegments,
			material,
			materials,
			intersection,
			media,
			direct_light_rays,
			direct_light_isects,
			lights,
			num_lights,
			geoms,
			rng,
			u01);
	}
}

// kernel to handle interactions within a surface (instead of medium)
__global__ void generateSurfaceDirectLightSample(
	int num_paths,
	int max_depth,
	PathSegment* pathSegments,
	Material* materials,
	ShadeableIntersection* intersections,
	Medium* media,
	MISLightRay* direct_light_rays,
	MISLightIntersection* direct_light_isects,
	Light* lights,
	int num_lights,
	Geom* geoms
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}
		if (intersections[idx].mi.medium >= 0) {
			return;
		}

		ShadeableIntersection intersection = intersections[idx];

		if (intersection.materialId < 0) {
			// Change ray direction
			pathSegments[idx].ray.origin = pathSegments[idx].ray.origin + (intersection.t * pathSegments[idx].ray.direction) + (0.001f * pathSegments[idx].ray.direction);
			pathSegments[idx].medium = glm::dot(pathSegments[idx].ray.direction, intersection.surfaceNormal) > 0 ? intersection.mediumInterface.outside :
			intersection.mediumInterface.inside;
			pathSegments[idx].remainingBounces--;
			pathSegments[idx].prev_hit_null_material = true;
			return;
		}

		Material material = materials[intersection.materialId];

		if (material.emittance > 0.0f) {
			if (pathSegments[idx].remainingBounces == max_depth || pathSegments[idx].prev_hit_was_specular) {
				// only color lights on first hit
				pathSegments[idx].accumulatedIrradiance += (material.R * material.emittance) * pathSegments[idx].rayThroughput;
			}
			pathSegments[idx].remainingBounces = 0;
			return;
		}

		pathSegments[idx].prev_hit_was_specular = material.type == SPEC_BRDF || material.type == SPEC_BTDF || material.type == SPEC_GLASS;

		if (pathSegments[idx].prev_hit_was_specular) {
			return;
		}

		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0, 1.0);

		// TODO: Check surface intersection bsdf doesn't exist
		computeDirectLightSamplePreVis(
			idx,
			pathSegments,
			material,
			materials,
			intersection,
			media,
			direct_light_rays,
			direct_light_isects,
			lights,
			num_lights,
			geoms,
			rng,
			u01);
	}
}


__global__ void computeVisVolumetric(
	int num_paths
	, PathSegment* pathSegments
	, MISLightRay* direct_light_rays
	, Geom* geoms
	, int geoms_size
	, Tri* tris
	, int tris_size
	, MISLightIntersection* direct_light_intersections
	, BVHNode_GPU* bvh_nodes
	, Medium* media,
	const nanovdb::NanoGrid<float>* media_density
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		if (pathSegments[path_index].remainingBounces == 0) {
			return;
		}
		else if (pathSegments[path_index].prev_hit_was_specular) {
			return;
		}
		else if (pathSegments[path_index].prev_hit_null_material) {
			return;
		}
		
		MISLightRay r = direct_light_rays[path_index];
		MISLightIntersection isect = direct_light_intersections[path_index];

		thrust::default_random_engine& rng = pathSegments[path_index].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
		
		glm::vec3 Tr = glm::vec3(1.0f);

		while (true) {
			// Surface Intersection
			float t_min = MAX_INTERSECT_DIST;
			int obj_ID = -1;
			float t;
			glm::vec3 tmp_normal;

#ifdef ENABLE_TRIS
			if (tris_size != 0) {
				int stack_pointer = 0;
				int cur_node_index = 0;
				int node_stack[32];
				BVHNode_GPU cur_node;
				glm::vec3 P;
				glm::vec3 s;
				float t1;
				float t2;
				float tmin;
				float tmax;
				while (true) {
					cur_node = bvh_nodes[cur_node_index];

					// (ray-aabb test node)
					t1 = (cur_node.AABB_min.x - r.ray.origin.x) * r.ray.direction_inv.x;
					t2 = (cur_node.AABB_max.x - r.ray.origin.x) * r.ray.direction_inv.x;

					tmin = glm::min(t1, t2);
					tmax = glm::max(t1, t2);

					t1 = (cur_node.AABB_min.y - r.ray.origin.y) * r.ray.direction_inv.y;
					t2 = (cur_node.AABB_max.y - r.ray.origin.y) * r.ray.direction_inv.y;

					tmin = glm::max(tmin, glm::min(t1, t2));
					tmax = glm::min(tmax, glm::max(t1, t2));

					t1 = (cur_node.AABB_min.z - r.ray.origin.z) * r.ray.direction_inv.z;
					t2 = (cur_node.AABB_max.z - r.ray.origin.z) * r.ray.direction_inv.z;

					tmin = glm::max(tmin, glm::min(t1, t2));
					tmax = glm::min(tmax, glm::max(t1, t2));

					if (tmax >= tmin) {
						// we intersected AABB
						if (cur_node.tri_index != -1) {
							// this is leaf node
							// triangle intersection test
							Tri tri = tris[cur_node.tri_index];


							t = glm::dot(tri.plane_normal, (tri.p0 - r.ray.origin)) / glm::dot(tri.plane_normal, r.ray.direction);
							if (t >= -0.0001f) {
								P = r.ray.origin + t * r.ray.direction;

								// barycentric coords
								s = glm::vec3(glm::length(glm::cross(P - tri.p1, P - tri.p2)),
									glm::length(glm::cross(P - tri.p2, P - tri.p0)),
									glm::length(glm::cross(P - tri.p0, P - tri.p1))) / tri.S;

								if (s.x >= -0.0001f && s.x <= 1.0001f && s.y >= -0.0001f && s.y <= 1.0001f &&
									s.z >= -0.0001f && s.z <= 1.0001f && (s.x + s.y + s.z <= 1.0001f) && (s.x + s.y + s.z >= -0.0001f) && t_min > t) {
									t_min = t;
									tmp_normal = glm::normalize(s.x * tri.n0 + s.y * tri.n1 + s.z * tri.n2);
									// Check if surface is medium transition
									if (IsMediumTransition(tri.mediumInterface)) {
										isect.mediumInterface = tri.mediumInterface;
									}
									else {
										isect.mediumInterface.inside = r.medium;
										isect.mediumInterface.outside = r.medium;
									}
								}
							}
							// if last node in tree, we are done
							if (stack_pointer == 0) {
								break;
							}
							// otherwise need to check rest of the things in the stack
							stack_pointer--;
							cur_node_index = node_stack[stack_pointer];
						}
						else {
							node_stack[stack_pointer] = cur_node.offset_to_second_child;
							stack_pointer++;
							cur_node_index++;
						}
					}
					else {
						// didn't intersect AABB, remove from stack
						if (stack_pointer == 0) {
							break;
						}
						stack_pointer--;
						cur_node_index = node_stack[stack_pointer];
					}
				}
			}
#endif

			for (int i = 0; i < geoms_size; ++i)
			{
				Geom& geom = geoms[i];


				if (geom.type == SPHERE) {
#ifdef ENABLE_SPHERES
					t = sphereIntersectionTest(geom, r.ray, tmp_normal);
#endif
				}
				else if (geom.type == SQUAREPLANE) {
#ifdef ENABLE_SQUAREPLANES
					t = squareplaneIntersectionTest(geom, r.ray, tmp_normal);
#endif
				}
				else {
#ifdef ENABLE_RECTS
					t = boxIntersectionTest(geom, r.ray, tmp_normal);
#endif
				}

				if (t_min > t)
				{
					t_min = t;
					obj_ID = i;

					// Check if surface is medium transition
					if (IsMediumTransition(geom.mediumInterface)) {
						isect.mediumInterface = geom.mediumInterface;
					}
					else {
						isect.mediumInterface.inside = r.medium;
						isect.mediumInterface.outside = r.medium;
					}
				}
			}


			// if intersected object is not a "invisible" bounding box, the ray is occluded
			if (obj_ID == -1 || (obj_ID != -1 && obj_ID != r.light_ID /* TODO: && !is_bounding_box */)) {
				direct_light_intersections[path_index].LTE = glm::vec3(0.0f, 0.0f, 0.0f);
				return;
			}

			// if the current ray has a medium, then attenuate throughput based on transmission and distance traveled
			if (r.medium != -1) {
				if (media[r.medium].type == HOMOGENEOUS) {
					Tr *= Tr_homogeneous(media[r.medium], r.ray, t_min);
				}
				else {
					Tr *= Tr_heterogeneous(media[r.medium], r, media_density, t_min, rng, u01);
				}
			}

			// if the intersected object IS the light source we selected, we are done
			if (obj_ID == r.light_ID) {
				direct_light_intersections[path_index].LTE *= Tr;
				return;
			}

			
			// We encountered a bounding box/entry/exit of a volume, so we must change our medium value, update the origin, and traverse again
			glm::vec3 old_origin = r.ray.origin;
			r.ray.origin = old_origin + (r.ray.direction * t_min);
			// TODO: maybe change ray direction

			r.medium = glm::dot(r.ray.direction, tmp_normal) > 0 ? isect.mediumInterface.outside :
				isect.mediumInterface.inside;
		}
	}
}

__global__ void mediumSpawnPathSegment(
	int iter
	, int num_paths
	, ShadeableIntersection* intersections
	, MISLightIntersection* direct_light_isects
	, int num_lights
	, PathSegment* pathSegments
	, Material* materials
	, Medium* media) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}
		else if (intersections[idx].mi.medium == -1) {
			return;
		}
		else if (pathSegments[idx].prev_hit_null_material) {
			pathSegments[idx].prev_hit_null_material = false;
			return;
		}

		ShadeableIntersection intersection = intersections[idx];

		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);


		pathSegments[idx].accumulatedIrradiance += pathSegments[idx].rayThroughput * direct_light_isects[idx].LTE; // TODO: * uniform sample one light;
		glm::vec3 wo = -pathSegments[idx].ray.direction;
		glm::vec3 wi;
		Sample_p(wo, &wi, glm::vec2(u01(rng), u01(rng)), media[pathSegments[idx].medium].g);


		// Create new ray
		pathSegments[idx].ray.direction = wi;
		pathSegments[idx].ray.direction_inv = 1.0f / wi;
		pathSegments[idx].ray.origin = intersection.mi.samplePoint + (wi * 0.001f);
		// TRY: Assert(mediumInterface.inside == mediumInterface.outside);
		//pathSegments[idx].medium = pathSegments[idx].medium;
		pathSegments[idx].medium = intersection.mi.medium;
		pathSegments[idx].remainingBounces--;
	}
}

__global__ void surfaceSpawnPathSegment(
	int iter
	, int num_paths
	, ShadeableIntersection* intersections
	, MISLightIntersection* direct_light_isects
	, int num_lights
	, PathSegment* pathSegments
	, Material* materials
	, Medium* media) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}
		if (intersections[idx].mi.medium >= 0) {
			return;
		}
		else if (pathSegments[idx].prev_hit_null_material) {
			pathSegments[idx].prev_hit_null_material = false;
			return;
		}

		ShadeableIntersection intersection = intersections[idx];
		MISLightIntersection direct_light_intersection = direct_light_isects[idx];

		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		thrust::uniform_real_distribution<float> u01(0.0, 1.0);

		Material m = materials[intersection.materialId];

		glm::vec3 intersect_point = pathSegments[idx].ray.origin + intersection.t * pathSegments[idx].ray.direction;

		if (!pathSegments[idx].prev_hit_was_specular) {
			pathSegments[idx].accumulatedIrradiance += pathSegments[idx].rayThroughput * direct_light_isects[idx].LTE; // TODO: * uniform sample one light;
		}
		glm::vec3 wi = glm::vec3(0.0f);
		glm::vec3 f = glm::vec3(0.0f);
		float pdf = 0.0f;
		float absDot = 0.0f;

		//thrust::uniform_real_distribution<float> u01(0, 1);

		// Physically based BSDF sampling influenced by PBRT
		// https://www.pbr-book.org/3ed-2018/Reflection_Models/Specular_Reflection_and_Transmission
		// https://www.pbr-book.org/3ed-2018/Reflection_Models/Lambertian_Reflection

		if (m.type == SPEC_BRDF) {
			wi = glm::reflect(pathSegments[idx].ray.direction, intersection.surfaceNormal);
			absDot = glm::abs(glm::dot(intersection.surfaceNormal, wi));
			pdf = 1.0f;
			if (absDot >= -0.0001f && absDot <= -0.0001f) {
				f = m.R;
			}
			else {
				f = m.R / absDot;
			}
		}
		else if (m.type == SPEC_BTDF) {
			// spec refl
			float eta = m.ior;
			if (glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction) < 0.0001f) {
				// outside
				eta = 1.0f / eta;
				wi = glm::refract(pathSegments[idx].ray.direction, intersection.surfaceNormal, eta);
			}
			else {
				// inside
				wi = glm::refract(pathSegments[idx].ray.direction, -intersection.surfaceNormal, eta);
			}
			absDot = glm::abs(glm::dot(intersection.surfaceNormal, wi));
			pdf = 1.0f;
			if (glm::length(wi) <= 0.0001f) {
				// total internal reflection
				f = glm::vec3(0.0f);
			}
			else if (absDot >= -0.0001f && absDot <= -0.0001f) {
				f = m.T;
			}
			else {
				f = m.T / absDot;
			}
		}
		else if (m.type == SPEC_GLASS) {
			// spec glass
			float eta = m.ior;
			if (u01(rng) < 0.5f) {
				// spec refl
				wi = glm::reflect(pathSegments[idx].ray.direction, intersection.surfaceNormal);
				absDot = glm::abs(glm::dot(intersection.surfaceNormal, wi));
				pdf = 1.0f;
				if (absDot == 0.0f) {
					f = m.R;
				}
				else {
					f = m.R / absDot;
				}
				f *= fresnelDielectric(glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction), m.ior);
			}
			else {
				// spec refr
				if (glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction) < 0.0f) {
					// outside
					eta = 1.0f / eta;
					wi = glm::refract(pathSegments[idx].ray.direction, intersection.surfaceNormal, eta);
				}
				else {
					// inside
					wi = glm::refract(pathSegments[idx].ray.direction, -intersection.surfaceNormal, eta);
				}
				absDot = glm::abs(glm::dot(intersection.surfaceNormal, wi));
				pdf = 1.0f;
				if (glm::length(wi) <= 0.0001f) {
					// total internal reflection
					f = glm::vec3(0.0f);
				}
				if (absDot == 0.0f) {
					f = m.T;
				}
				else {
					f = m.T / absDot;
				}
				f *= glm::vec3(1.0f) - fresnelDielectric(glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction), m.ior);
			}
			f *= 2.0f;
		}
		else {
			// diffuse
			wi = glm::normalize(calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng, u01));
			if (m.type == DIFFUSE_BTDF) {
				wi = -wi;
			}
			absDot = glm::abs(glm::dot(intersection.surfaceNormal, wi));
			pdf = absDot * 0.31831f;
			f = m.R * 0.31831f;
		}

		pathSegments[idx].rayThroughput *= f * absDot / pdf;

		// Change ray direction
		pathSegments[idx].ray.direction = wi;
		pathSegments[idx].ray.direction_inv = 1.0f / wi;
		pathSegments[idx].ray.origin = intersect_point + (wi * 0.001f);
		//pathSegments[idx].medium = intersection.mi.medium;
		pathSegments[idx].medium = glm::dot(pathSegments[idx].ray.direction, intersection.surfaceNormal) > 0 ? intersection.mediumInterface.outside :
			intersection.mediumInterface.inside;
		pathSegments[idx].remainingBounces--;
	}
}

__global__ void russianRouletteKernel_Vol(int iter, int num_paths, PathSegment* pathSegments)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0) {
			return;
		}
		thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
		thrust::default_random_engine& rng = pathSegments[idx].rng_engine;
		float random_num = u01(rng);
		float max_channel = glm::max(glm::max(pathSegments[idx].rayThroughput.r, pathSegments[idx].rayThroughput.g), pathSegments[idx].rayThroughput.b);
		if (max_channel < random_num) {
			pathSegments[idx].remainingBounces = 0;
		}
		else {
			pathSegments[idx].rayThroughput /= max_channel;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather_Vol(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[index] += iterationPath.accumulatedIrradiance;
	}
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO_Vol(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		pix /= iter;

		// reinhard (HDR)
		pix /= (pix + glm::vec3(1.0f));

		// gamma correction
		pix = glm::pow(pix, glm::vec3(0.454545f));

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}


struct is_done
{
	__host__ __device__
		bool operator()(const PathSegment &path)
	{
		return path.remainingBounces != 0;
	}
};

struct material_sort
{
	__host__ __device__
		bool operator()(const ShadeableIntersection& isect_0, const ShadeableIntersection& isect_1)
	{
		return isect_0.materialId < isect_1.materialId;
	}
};

void volPathtrace(uchar4* pbo, int frame, int iter) {

	//std::cout << "============================== " << iter << " ==============================" << std::endl;

	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + BLOCK_SIZE_2D - 1) / BLOCK_SIZE_2D,
		(cam.resolution.y + BLOCK_SIZE_2D - 1) / BLOCK_SIZE_2D);


	// 1D block for path tracing
	const int blockSize1d = BLOCK_SIZE_1D;

	int depth = 0;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;

	dim3 numblocksPathSegmentTracing = (pixelcount_vol + blockSize1d - 1) / blockSize1d;

	// gen ray
	thrust::default_random_engine rng = makeSeededRandomEngine_Vol(iter, iter, iter);
	thrust::uniform_real_distribution<float> upixel(0.0, 1.0f);

	float jitterX = upixel(rng);
	float jitterY = upixel(rng);

	generateRayFromCamera_Vol << <blocksPerGrid2d, blockSize2d >> > (cam,
		iter, traceDepth, jitterX, jitterY, dev_paths);

	while (!iterationComplete) {
		//std::cout << "depth: " << depth << std::endl;
		// When intersecting with primitive, determine if there is a medium transition or not
		// Update isect struct's mediumInterface member variable with the appropriate mediumInterface
		computeIntersections_Vol << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, pixelcount_vol
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_tris
			, hst_scene->num_tris
			, dev_intersections
			, dev_bvh_nodes
			);

		depth++;
		
		// Attenuating ray throughput with medium stuff (phase function)
		// Check if throughput is black, and break out of loop (set remainingBounces to 0)
		sampleParticipatingMedium << <numblocksPathSegmentTracing, blockSize1d >> > (
			pixelcount_vol,
			dev_paths,
			dev_intersections,
			dev_media,
			dev_media_density);
		
		// If medium interaction is valid, then sample light and pick new direction by sampling phase function distribution
		// Else, handle surface interaction
		generateMediumDirectLightSample << < numblocksPathSegmentTracing, blockSize1d >> > (
			pixelcount_vol,
			traceDepth,
			dev_paths,
			dev_materials,
			dev_intersections,
			dev_media,
			dev_direct_light_rays,
			dev_direct_light_isects,
			dev_lights,
			hst_scene->lights.size(),
			dev_geoms);

		generateSurfaceDirectLightSample << < numblocksPathSegmentTracing, blockSize1d >> > (
			pixelcount_vol,
			traceDepth,
			dev_paths,
			dev_materials,
			dev_intersections,
			dev_media,
			dev_direct_light_rays,
			dev_direct_light_isects,
			dev_lights,
			hst_scene->lights.size(),
			dev_geoms);

		computeVisVolumetric << < numblocksPathSegmentTracing, blockSize1d >> > (
			pixelcount_vol,
			dev_paths,
			dev_direct_light_rays,
			dev_geoms,
			hst_scene->geoms.size(),
			dev_tris,
			hst_scene->num_tris,
			dev_direct_light_isects,
			dev_bvh_nodes,
			dev_media,
			dev_media_density
			);
				
		mediumSpawnPathSegment << < numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			pixelcount_vol,
			dev_intersections,
			dev_direct_light_isects,
			hst_scene->lights.size(),
			dev_paths,
			dev_materials,
			dev_media);

		surfaceSpawnPathSegment << < numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			pixelcount_vol,
			dev_intersections,
			dev_direct_light_isects,
			hst_scene->lights.size(),
			dev_paths,
			dev_materials,
			dev_media);
		
		// RUSSIAN ROULETTE
		/*if (depth >= 5)
		{
			russianRouletteKernel_Vol << <numblocksPathSegmentTracing, blockSize1d >> > (
				iter,
				pixelcount_vol,
				dev_paths
				);
		}*/

		if (depth == traceDepth) { iterationComplete = true; }

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}


	// Assemble this iteration and apply it to the image
	finalGather_Vol << <numblocksPathSegmentTracing, blockSize1d >> > (pixelcount_vol, dev_image, dev_paths);

	// Send results to OpenGL buffer for rendering
	sendImageToPBO_Vol << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount_vol * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}